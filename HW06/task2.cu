#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <ctime>
#include <omp.h>
#include <random>
#include <hiprand/hiprand_kernel.h>
#include "stencil.cuh"

void stencil(const float* image, const float* output, float* mask, unsigned int n, unsigned int R, unsigned int threads_per_block) {
    int blocks = (n + threads_per_block - 1) / threads_per_block;
    int shared_mem_size = (threads_per_block + 2 * R) * sizeof(float);
    stencil_kernel<<<blocks, threads_per_block, shared_mem_size>>>(image, output, mask, n, R);
}

__global__ void stencil_kernel(const float* image, const float* mask, float* output, unsigned int n, unsigned int R, unsigned int threads_per_block) {
    extern __shared__ float shared[];

    int tid = threadIdx.x;
    int gid = blockIdx.x * blockDim.x + tid;

    int shared_index = tid + R;

    if (gid < n) {
        shared[shared_index] = image[gid];
    } else {
        shared[shared_index] = 1.0f;
    }

    if (tid < R) {
        int left_idx = gid - R;
        shared[tid] = (left_idx < 0) ? 1.0f : image[left_idx];
    }

    if (tid >= blockDim.x - R) {
        int right_idx = gid + R;
        shared[shared_index + R] = (right_idx >= n) ? 1.0f : image[right_idx];
    }

    __syncthreads();

    if (gid < n) {
        float sum = 0.0f;
        for (int j = -R; j <= R; j++) {
            sum += shared[shared_index + j] * mask[j + R];
        }
        output[gid] = sum;
    }
}

int main(int argc, char* argv[]) {
    if (argc < 2) {
        std::cerr << "Usage: " << argv[0] << " <matrix_size>\n";
        return 1;
    }

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    int n = std::atoi(argv[1]);
    int R = std::atoi(argv[2]);
    unsigned int threads_per_block = std::atoi(argv[3]);
    int total_size = n * n;
    size_t bytes = total_size * sizeof(float);
    int mask_len = 2 * R + 1;

    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<float> dist(-1.0f, 1.0f);

    // Host allocations
    float* h_image = new float[n];
    float* h_mask = new float[mask_len];
    float* h_output = new float[n];

<<<<<<< HEAD
    for (:_t i = 0; i < size; ++i) {
=======
    for (int i = 0; i < n; ++i) {
>>>>>>> 8096a76 (Euler task2 debug)
        h_image[i] = dist(gen);
        h_mask[i] = dist(gen);
    }    

    // Device allocations
    float *image, *output, *mask;
    hipMalloc(&image, n * sizeof(float));
    hipMalloc(&output, n * sizeof(float));
    hipMalloc(&mask, mask_len * sizeof(float));

    hipMemcpy(image, h_image, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(mask, h_mask, mask_len * sizeof(float), hipMemcpyHostToDevice);

    stencil(image, output, mask, n, R, threads_per_block);
    hipDeviceSynchronize();

    hipMemcpy(h_output, output, n * sizeof(float), hipMemcpyDeviceToHost);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    printf("%f\n", h_output[n]);
    printf("%f\n", milliseconds);


    // Cleanup
    delete[] h_image;
    delete[] h_output;
    delete[] h_mask;
    hipFree(image);
    hipFree(output);
    hipFree(mask);

    return 0;
}
