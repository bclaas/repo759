#include <cstdio>
#include <cstdlib>
#include <ctime>
#include <hip/hip_runtime.h>
#define N 16

__global__ void computeKernel(int *dA, int a){
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N) {
        dA[idx] = a * threadIdx.x + blockIdx.x;
    }
}

int main(){
    srand(time(NULL));
    int a = rand() % 10 + 1;
    int *dA, hA[N];
    hipMalloc((void**)&dA, N * sizeof(int));
    computeKernel<<<2, 8>>>(dA, a);
    hipMemcpy(hA, dA, N * sizeof(int), hipMemcpyDeviceToHost);
    for (int i = 0; i < N; i++) { printf("%d ", hA[i]); }
    printf("\n");
    hipFree(dA);
    return 0;
}

