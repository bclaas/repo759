#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <ctime>
#include "vscale.cuh"

void randomizeArray(float *arr, int n, float min, float max) {
    for (int i = 0; i < n; i++) {
        arr[i] = min + static_cast<float>(rand()) / (static_cast<float>(RAND_MAX / (max - min)));
    }
}

int main(int argc, char **argv) {
    if (argc != 2) {
        printf("Usage: %s <n>\n", argv[0]);
        return 1;
    }
    
    int n = atoi(argv[1]);
    srand(time(NULL));
    
    float *h_a = (float*)malloc(n * sizeof(float));
    float *h_b = (float*)malloc(n * sizeof(float));
    randomizeArray(h_a, n, -10.0f, 10.0f);
    randomizeArray(h_b, n, 0.0f, 1.0f);
    
    float *d_a, *d_b;
    hipMalloc((void**)&d_a, n * sizeof(float));
    hipMalloc((void**)&d_b, n * sizeof(float));
    hipMemcpy(d_a, h_a, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, n * sizeof(float), hipMemcpyHostToDevice);
    
    int threadsPerBlock = 16;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    
    vscale<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, n);
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    
    hipMemcpy(h_b, d_b, n * sizeof(float), hipMemcpyDeviceToHost);
    
    printf("%f\n", milliseconds);
    printf("%f\n", h_b[0]);
    printf("%f\n", h_b[n - 1]);
    
    hipFree(d_a);
    hipFree(d_b);
    free(h_a);
    free(h_b);
    
    return 0;
}
