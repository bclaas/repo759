#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <cstdlib>
#include <ctime>
#include "reduce.cuh"

int main(int argc, char** argv) {
    if (argc != 3) {
        std::cerr << "Usage: " << argv[0] << " <N> <threads_per_block>\n";
        return 1;
    }

    size_t N = atoi(argv[1]);
    unsigned int threads_per_block = atoi(argv[2]);

    // Allocate and initialize host array with random values in range [-1, 1]
    std::vector<float> h_data(N);
    srand(time(NULL));
    for (size_t i = 0; i < N; ++i) {
        h_data[i] = (float(rand()) / RAND_MAX) * 2.0f - 1.0f; // [-1, 1]
    }

    float* d_in;
    float* d_out;
    hipMalloc(&d_in, N * sizeof(float));
    size_t blocks = (N + threads_per_block * 2 - 1) / (threads_per_block * 2);
    hipMalloc(&d_out, blocks * sizeof(float));

    hipMemcpy(d_in, h_data.data(), N * sizeof(float), hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    reduce(&d_in, &d_out, N, threads_per_block);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // Copy the result back to host
    float h_result;
    hipMemcpy(&h_result, d_in, sizeof(float), hipMemcpyDeviceToHost);

    float ms = 0;
    hipEventElapsedTime(&ms, start, stop);

    std::cout << "Sum: " << h_result << "\n";
    std::cout << "Time: " << ms << " ms\n";

    // Clean up
    hipFree(d_in);
    hipFree(d_out);

    return 0;
}
